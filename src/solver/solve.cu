#include "hip/hip_runtime.h"
#include <unistd.h>
#include <iostream>

#include "parameters.h"
#include "structs.h"
#include "EOS.h"
#include "init.h"
#include "orbit.h"
#include "planet.h"

__device__ int glo_idx(int i, int j, int k, int imax, int jmax)
{
	return i + imax*(j+jmax*k);
}

__device__ int jlim(int j, int jmax)
{
	j = j%jmax;
	if (j<0) j += jmax;
	return j;
}

__device__ double get_dv_dr_dev(int geom, double ra, double dr)
{
	if 	(geom==5)
	{
		if (dr<1.0e-4) return sin(ra+0.5*dr);
		else           return (cos(ra)-cos(ra+dr))/dr;
	}
	//else if (geom==4) return 1.0;	
	//else if (geom==3) return 1.0;
	else if (geom==2) return ra*ra + ra*dr + dr*dr/3.0;
	else if (geom==1) return ra + dr/2.0;
	else return 1.0;
}

__device__ double get_volume_dev(int geom, double ra, double dr)
{
	return dr*get_dv_dr_dev(geom,ra,dr);
}

void syncallstreams(Grid* dev)
{
	for (int n=0; n<ndev; n++) hipStreamSynchronize(dev[n].stream);
}

#include "cool/cool.cu"
#include "kill/killwave.cu"
#include "force/forces.cu"
#include "recon/recon.cu"
#include "boundary/boundary.cu"

#include "force/viscosity.cu"
#include "riemann/riemann.cu"
#include "advection/advection.cu"

__global__ void clear_flux(int mx, int my, int mz, Cell* F)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i<mx*my*mz) F[i].zero();
	return;
}

__global__ void clear_forces(int mx, int my, int mz, double* fx, double* fy, double* fz)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i<mx*my*mz) 
	{
		fx[i] = 0.0;
		fy[i] = 0.0;
		fz[i] = 0.0;
	}
	return;
}

__global__ void sweepx(Grid G, Cell* C, double dt)
{
	__shared__ double xa[x_xthd+1], dx[x_xthd], xv[x_xthd];
	__shared__ double r[x_xthd*x_ydiv], p[x_xthd*x_ydiv], u[x_xthd*x_ydiv], v[x_xthd*x_ydiv], w[x_xthd*x_ydiv];
	double force;

	int i = threadIdx.x;
	int idx = i + blockIdx.x*x_xdiv;

	int j = threadIdx.y;
	int idy = j + blockIdx.y*x_ydiv + ypad;

	int k = threadIdx.z;
	int idz = k + blockIdx.z*x_zdiv + zpad;

	int ind = G.get_ind(idx,idy,idz);

	if (j==0)
	{
		xa[i] = G.get_xa(idx);
		if (i==blockDim.x-1) xa[i+1] = G.get_xa(idx+1);
		xv[i] = G.get_xv(idx);
	}
	__syncthreads();

	r[i+x_xthd*j] = C[ind].r;
	p[i+x_xthd*j] = C[ind].p;
	u[i+x_xthd*j] = C[ind].u;
	v[i+x_xthd*j] = C[ind].v;
	w[i+x_xthd*j] = C[ind].w;
	if (i>0) force = 0.5*(G.fx[ind] + G.fx[G.get_ind(idx-1,idy,idz)]);

	__syncthreads();

	if (j==0) dx[i] = xa[i+1] - xa[i];

	double rad = 0.5*(xa[i+1]+xa[i]);
	#if geomx == 1
	v[i+x_xthd*j] *= rad;
	#elif geomx == 2
	double rad_cyl = rad * sin(G.get_zc(idz));
	v[i+x_xthd*j] *= rad_cyl;
	w[i+x_xthd*j] *= rad;
	#endif
	__syncthreads();

//if (idy==2) printf("%f %f %f\n",rad,r[y_ythd*j+i],p[y_ythd*j+i]);

	/////////////////////////////////////////////////////
	Cell Del;
	Del =   riemann(geomx, xa, dx, xv, rad, &r[x_xthd*j], &p[x_xthd*j], &u[x_xthd*j], &v[x_xthd*j], &w[x_xthd*j], force, dt);
//if (idy==ypad && i>=xpad && i<x_xthd-xpad && idx>270 && idx<290) printf("%f %f: %f, %f, %e, %f, %f, %e\n",xa[i],xv[i+1]-xv[i],r[i+x_xthd*j],p[i+x_xthd*j],u[i+x_xthd*j],force,G.get_rot(idx,idz),Del.r);

	Del.multiply(G.get_yv(idy)*G.get_zv(idz));

	if (i>=xpad && i<x_xthd-xpad)
	{
		#if geomx == 1
		Del.v /= rad;
		#elif geomx == 2
		Del.v /= rad_cyl;
		Del.w /= rad;
		#endif
		G.F[ind].add(Del);
	}
	//if (idx==12 && G.get_j_shf(idx,idy,idz)==17) printf("sweepx: rad=%f azi=%f\n",G.get_xc(idx),G.get_yc(idy));

	return;
}


__global__ void sweepy(Grid G, Cell* C, double dt)
{
	__shared__ double ya[y_ythd+1], dy[y_ythd], yv[y_ythd];
	__shared__ double r[y_ythd*y_xdiv], p[y_ythd*y_xdiv], u[y_ythd*y_xdiv], v[y_ythd*y_xdiv], w[y_ythd*y_xdiv];
	double force;

	int i = threadIdx.x;
	int idy = i + blockIdx.x*y_ydiv;

	int j = threadIdx.y;
	int idx = j + blockIdx.y*y_xdiv + xpad;

	int k = threadIdx.z;
	int idz = k + blockIdx.z*x_zdiv + zpad;

	int ind = idx + G.xarr*idy + G.xarr*G.yarr*idz;

	if (j==0)
	{
		ya[i] = G.get_ya(idy);
		if (i==blockDim.x-1) ya[i+1] = G.get_ya(idy+1);
		yv[i] = G.get_yv(idy);
	}
	__syncthreads();

	if (j==0) dy[i] = ya[i+1] - ya[i];

	double rad;	
	#if geomy == 3
	rad = G.get_xc(idx);
	#elif geomy == 4
	rad = G.get_xc(idx);
	rad *= sin(G.get_zc(idz));
	#else
	rad = 1.0;
	#endif

	r[i+y_ythd*j] = C[ind].r;
	p[i+y_ythd*j] = C[ind].p;
	u[i+y_ythd*j] = C[ind].v - G.get_rot(idx,idz);
	v[i+y_ythd*j] = C[ind].w;
	w[i+y_ythd*j] = C[ind].u;

	#if geomy == 3 || geomy == 4
	u[i+y_ythd*j] -= rad*frame_omega;
	#endif

	if (i>0) force = 0.5*(G.fy[ind] + G.fy[G.get_ind(idx,idy-1,idz)]);
	__syncthreads();

	/////////////////////////////////////////////////////

	Cell Del;
	Del = riemann(geomy, ya, dy, yv, rad, &r[y_ythd*j], &p[y_ythd*j], &u[y_ythd*j], &v[y_ythd*j], &w[y_ythd*j], force, dt);

	Del.multiply(G.get_xv(idx)*G.get_zv(idz));

	#if geomy > 2
	Del.multiply(1.0/rad);
	#endif

	if (i>=ypad && i<y_ythd-ypad)
	{
		G.F[ind].r += Del.r;
		G.F[ind].p += Del.p;
		G.F[ind].u += Del.w;
		#if geomy == 3 || geomy == 4
		G.F[ind].v += Del.u + (G.get_rot(idx,idz) + rad*frame_omega)*Del.r;
		#else
		G.F[ind].v += Del.u + G.get_rot(idx,idz)*Del.r;
		#endif
		G.F[ind].w += Del.v;
	}

	return;
}

__global__ void sweepz(Grid G, Cell* C, double dt)
{
	__shared__ double za[z_zthd+1], dz[z_zthd], zv[z_zthd];
	__shared__ double r[z_zthd*z_xdiv], p[z_zthd*z_xdiv], u[z_zthd*z_xdiv], v[z_zthd*z_xdiv], w[z_zthd*z_xdiv];
	double force;

	int i = threadIdx.x;
	int idz = i + blockIdx.x*z_zdiv;

	int j = threadIdx.y;
	int idx = j + blockIdx.y*z_xdiv + xpad;

	int k = threadIdx.z;
	int idy = k + blockIdx.z*z_ydiv + ypad;

	int ind = G.get_ind(idx,idy,idz);

	if (j==0)
	{
		za[i] = G.get_za(idz);
		if (i==blockDim.x-1) za[i+1] = G.get_za(idz+1);
		zv[i] = G.get_zv(idz);
	}
	__syncthreads();

	if (j==0) dz[i] = za[i+1] - za[i];
	
	r[i+z_zthd*j] = C[ind].r;
	p[i+z_zthd*j] = C[ind].p;
	u[i+z_zthd*j] = C[ind].w;
	v[i+z_zthd*j] = C[ind].u;
	w[i+z_zthd*j] = C[ind].v;
	if (i>0) force = 0.5*(G.fz[ind] + G.fz[G.get_ind(idx,idy,idz-1)]);
	__syncthreads();

	double rad;
	rad = G.get_xc(idx);
	#if geomz == 5
	double rad_cyl;
	rad_cyl = rad * sin(0.5*(za[i+1]+za[i]));
	w[i+z_zthd*j] *= rad_cyl;
	#endif
	__syncthreads();

	/////////////////////////////////////////////////////

	Cell Del;
	Del = riemann(geomz, za, dz, zv, rad, &r[z_zthd*j], &p[z_zthd*j], &u[z_zthd*j], &v[z_zthd*j], &w[z_zthd*j], force, dt);
	Del.multiply(G.get_xv(idx)*G.get_yv(idy));

	#if geomz > 2
	Del.multiply(1.0/rad);
	#endif

	if (i>=zpad && i<z_zthd-zpad)
	{
		#if geomz == 5
		Del.w /= rad_cyl;
		#endif
		G.F[ind].r += Del.r;
		G.F[ind].p += Del.p;
		G.F[ind].u += Del.v;
		G.F[ind].v += Del.w;
		G.F[ind].w += Del.u;
	}

	return;
}

__global__ void update(Grid G, Cell* in, Cell* out, double dt, double div=1.0)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x + xpad;
	int j = threadIdx.y + blockIdx.y*blockDim.y + ypad;
	int k = threadIdx.z + blockIdx.z*blockDim.z + zpad;
	double vol;
	Cell Q;
	Cell D;
	int ind;

	double fx;
	#if ndim>1
	double fy;
	#endif
	#if ndim>2
	double fz;
	#endif

	if (i>=xpad && i<G.xarr-xpad)
	if (j>=ypad && j<G.yarr-ypad)
	if (k>=zpad && k<G.zarr-zpad)
	{
		ind = i + G.xarr*j + G.xarr*G.yarr*k;
		vol = G.get_xv(i)*G.get_yv(j)*G.get_zv(k);

		Q.copy(in[ind]);
		D.copy(G.F[ind]);
		D.multiply(div*dt);

		if (isnan(D.r)) printf("Error: update, %f, %f\n %e, %e, %e, %e, %e\n %e, %e, %e, %e, %e\n",G.get_xc(i),G.get_yc(j),Q.r,Q.p,Q.u,Q.v,Q.w,D.r,D.p,D.u,D.v,D.w);

		Q.p = get_energy(Q.r,Q.p,Q.u,Q.v,Q.w);
		Q.r *= vol;
		Q.p *= Q.r;
		Q.u *= Q.r;
		Q.v *= Q.r;
		Q.w *= Q.r;
		
		fx = G.fx[ind];
		Q.u += 0.5*Q.r*fx*dt;
		#if ndim>1
		fy = G.fy[ind];
		Q.v += 0.5*Q.r*fy*dt;
		#endif
		#if ndim>2
		fz = G.fz[ind];
		Q.w += 0.5*Q.r*fz*dt;
		#endif

		Q.add(D);

		Q.u += 0.5*Q.r*fx*dt;
		#if ndim>1
		Q.v += 0.5*Q.r*fy*dt;
		#endif
		#if ndim>2
		Q.w += 0.5*Q.r*fz*dt;
		#endif

		Q.u /= Q.r;
		Q.v /= Q.r;
		Q.w /= Q.r;
		Q.r /= vol;

		if (Q.r<=0.0)
		{
			Q.r = smallr;
			Q.p = get_cs2(G.get_xc(i),G.get_yc(j),G.get_zc(k))*Q.r;
			Q.u = in[ind].u;
			Q.v = in[ind].v;
			Q.w = in[ind].w;
			printf("Error: negative density at %f %f %f\n",G.get_xc(i),G.get_yc(j),G.get_zc(k));
		}
		else if (Q.p<=0.0)
		{
			Q.p = get_cs2(G.get_xc(i),G.get_yc(j),G.get_zc(k))*Q.r;
			printf("Error: negative pressure at %f %f %f\n",G.get_xc(i),G.get_yc(j),G.get_zc(k));
		}
		else
		{
			#if EOS_flag == 2
			#if internal_e_flag==0
			Q.p = fmax(Q.p*gamm/vol - gamm*Q.r*(Q.u*Q.u+Q.v*Q.v+Q.w*Q.w)/2.0,smallp);
			#else
			Q.p = fmax(Q.p*gamm/vol,smallp);
			#endif
			#elif EOS_flag == 0
			Q.p = get_cs2(G.get_xc(i),G.get_yc(j),G.get_zc(k))*Q.r;
			#endif
		}

		out[ind].copy(Q);
	}

	return;
}

__global__ void apply_viscosity(Grid G, Cell* C, double dt)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x + xpad;
	int j = threadIdx.y + blockIdx.y*blockDim.y + ypad;
	int k = threadIdx.z + blockIdx.z*blockDim.z + zpad;

	Cell T;
	double fx;
	#if ndim > 1
	double fy;
	#endif
	#if ndim > 2
	double fz;
	#endif
	int ind;

	if (i>=xpad && i<G.xarr-xpad)
	if (j>=ypad && j<G.yarr-ypad)
	if (k>=zpad && k<G.zarr-zpad)
	{		
		ind = G.get_ind(i,j,k);
		T.copy(C[ind]);

		fx = viscous_fx(G, C, i, j, k)/T.r;
		T.u += fx*dt;
		#if ndim > 1
		fy = viscous_fy(G, C, i, j, k)/T.r;
		T.v += fy*dt;
		#endif
		#if ndim > 2
		fz = viscous_fz(G, C, i, j, k)/T.r;
		T.w += fz*dt;
		#endif

		C[ind].copy(T);
	}

	return;
}

__global__ void compute_forces(Grid G, Cell* C, double dt, double x_dt, bool ave=false)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int k = threadIdx.z + blockIdx.z*blockDim.z;

	Cell T;
	double xc,yc,zc;
	double fx = 0.0;
	double fy = 0.0;
	double fz = 0.0;
	int ind;

	if (i>=xpad-1 && i<G.xarr-xpad+1)
	if (j>=ypad-1 && j<G.yarr-ypad+1)
	if (k>=zpad-1 && k<G.zarr-zpad+1)
	{		
		ind = G.get_ind(i,j,k);
		T.copy(C[ind]);

		xc = G.get_xc(i);
		#if ndim > 1
		yc = G.get_yc(j) + G.get_rot(i,k)*x_dt;
		#else
		yc = 0.0;
		#endif
		#if ndim > 2
		zc = G.get_zc(k);
		#else
		zc = 0.0;
		#endif

		#if ndim > 1
		fy = get_fy(xc,yc,zc,T.u,T.v,T.w,G.planets);
		#ifdef visc_flag
		fy += viscous_fy(G, C, i, j, k)/T.r;
		#endif
		if (ave) G.fy[ind] = (G.fy[ind] + fy)/2.0;
		else     G.fy[ind] = fy;
		#endif

		#if ndim > 2
		fz = get_fz(xc,yc,zc,T.u,T.v+0.5*dt*fy,T.w,G.planets);
		#ifdef visc_flag
		fz += viscous_fz(G, C, i, j, k)/T.r;
		#endif
		if (ave) G.fz[ind] = (G.fz[ind] + fz)/2.0;
		else     G.fz[ind] = fy;
		#endif

		fx = get_fx(xc,yc,zc,T.u,T.v+0.5*dt*fy,T.w+0.5*dt*fz,G.planets);
		#ifdef visc_flag
		fx += viscous_fx(G, C, i, j, k)/T.r;
		#endif
		if (ave) G.fx[ind] = (G.fx[ind] + fx)/2.0;
		else     G.fx[ind] = fx;

	}

	return;
}

__global__ void apply_forces(Grid G, Cell* C, double dt)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x + xpad;
	int j = threadIdx.y + blockIdx.y*blockDim.y + ypad;
	int k = threadIdx.z + blockIdx.z*blockDim.z + zpad;

	Cell T;
	int ind;

	if (i>=xpad && i<G.xarr-xpad)
	if (j>=ypad && j<G.yarr-ypad)
	if (k>=zpad && k<G.zarr-zpad)
	{		
		ind = G.get_ind(i,j,k);
		T.copy(C[ind]);
/*
		xc = G.get_xc(i);
		#if ndim > 1
		yc = G.get_yc(j) + G.get_rot(i,k)*0.5*dt;
		#else
		yc = 0.0;
		#endif
		#if ndim > 2
		zc = G.get_zc(k);
		#else
		zc = 0.0;
		#endif

		#if ndim > 1
		fy = get_fy(xc,yc,zc,T.u,T.v,T.w,G.planets);
		#ifdef visc_flag
		fy += viscous_fy(G, C, i, j, k)/T.r;
		#endif
		G.fy[ind] = fy;
		#endif

		#if ndim > 2
		fz = get_fz(xc,yc,zc,T.u,T.v+0.5*dt*fy,T.w,G.planets);
		#ifdef visc_flag
		fz += viscous_fz(G, C, i, j, k)/T.r;
		#endif
		G.fz[ind] = fz;
		#endif

		fx = get_fx(xc,yc,zc,T.u,T.v+0.5*dt*fy,T.w+0.5*dt*fz,G.planets);
		#ifdef visc_flag
		fx += viscous_fx(G, C, i, j, k)/T.r;
		#endif
		G.fx[ind] = fx;
*/
		T.u += G.fx[ind]*dt;

		#if ndim > 1
		T.v += G.fy[ind]*dt;
		#endif

		#if ndim > 2
		T.w += G.fz[ind]*dt;
		#endif

		C[ind].copy(T);
	}

	return;
}

void DS(Grid* dev, double time, double dt)
{
	int nx, ny, nz;
	int mx, my, mz;
	int bsz = 64;

	//////////////////////////////////////////////////////////////

	#ifdef visc_flag
	syncallstreams(dev);
	viscosity_tensor_evaluation1(dev);
	#endif

	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		mx = dev[n].xarr;
		my = dev[n].yarr;
		mz = dev[n].zarr;

		compute_forces<<< dim3((mx+bsz-1)/bsz,my,mz), bsz, 0, dev[n].stream >>> (dev[n], dev[n].C, 0.0, 0.0);
	}

	//////////////////////////////////////////////////////////////

	syncallstreams(dev);

	boundx(dev);
	#if ndim>1
	boundy(dev);
	#endif
	#if ndim>2
	boundz(dev);
	#endif

	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		nx = dev[n].xres;
		ny = dev[n].yres;
		nz = dev[n].zres;

		mx = dev[n].xarr;
		my = dev[n].yarr;
		mz = dev[n].zarr;

		clear_flux<<< (mx*my*mz+bsz-1)/bsz, bsz, 0, dev[n].stream >>>(mx, my, mz, dev[n].F);

		sweepx<<< dim3(nx/x_xdiv,ny/x_ydiv,nz/x_zdiv), dim3(x_xthd,x_ydiv,x_zdiv), 2*sizeof(double)*x_xthd*x_ydiv*x_zdiv, dev[n].stream >>>
		      (dev[n], dev[n].C, dt);

		#if ndim>1
		sweepy<<< dim3(ny/y_ydiv,nx/y_xdiv,nz/y_zdiv), dim3(y_ythd,y_xdiv,y_zdiv), 2*sizeof(double)*y_ythd*y_xdiv*y_zdiv, dev[n].stream >>>
		      (dev[n], dev[n].C, dt);
		#endif

		#if ndim>2
		sweepz<<< dim3(nz/z_zdiv,nx/z_xdiv,ny/z_ydiv), dim3(z_zthd,z_xdiv,z_ydiv), 2*sizeof(double)*z_zthd*z_xdiv*z_ydiv, dev[n].stream >>>
		      (dev[n], dev[n].C, dt);
		#endif
	}

	//////////////////////////////////////////////////////////////

	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		nx = dev[n].xres;
		ny = dev[n].yres;
		nz = dev[n].zres;

		update<<< dim3(nx/x_xdiv,ny,nz), x_xthd, 0, dev[n].stream >>> (dev[n], dev[n].C, dev[n].T, dt);
	}

	//////////////////////////////////////////////////////////////

	evolve_planet(dev,time+dt,dt);

	//////////////////////////////////////////////////////////////

	#ifdef visc_flag
	syncallstreams(dev);
	viscosity_tensor_evaluation2(dev);
	#endif

	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		mx = dev[n].xarr;
		my = dev[n].yarr;
		mz = dev[n].zarr;

		compute_forces<<< dim3((mx+bsz-1)/bsz,my,mz), bsz, 0, dev[n].stream >>> (dev[n], dev[n].T, 0.0, dt, true);
	}

	//////////////////////////////////////////////////////////////

	for (int n=0; n<ndev; n++)
	{
		hipSetDevice(n);

		nx = dev[n].xres;
		ny = dev[n].yres;
		nz = dev[n].zres;

		update<<< dim3(nx/x_xdiv,ny,nz), x_xthd, 0, dev[n].stream >>> (dev[n], dev[n].C, dev[n].C, dt);
	}

	//////////////////////////////////////////////////////////////

	#ifdef kill_flag
	killwave(dev, dt);
	#endif

	return;
}

void solve(Grid* dev, double time, double dt)
{
	#ifndef advec_flag

		#ifdef cool_flag
		cooling(dev, 0.5*dt);
		#endif

		#ifdef RadPres_flag
		compute_extinction(dev, 1.0);
		#endif

		#if mode_flag == 0
		DS(dev,time,dt);
		#elif mode_flag == 1
		#endif

		#ifdef OrbAdv_flag
		set_OrbAdv(dev,dt);
		shift_OrbAdv(dev);
		advecty(dev,dt);
		#endif

		#ifdef cool_flag
		cooling(dev, 0.5*dt);
		#endif
	#else

		advectx(dev,dt);

		#if ndim>1
		advecty(dev,dt);
		#endif

	#endif

	return;
}

