#include "hip/hip_runtime.h"

__device__ double get_g_apprx(double x, double k)
{
/*
	double n = 1.0;
	double tmp = -(k+1.0)*y/2.0;
	double sum = 1.0 + tmp;

	do
	{
		tmp *= -(k-n)*y/(n+2.0);
		sum += tmp;
		n += 1.0;
	} while (fabs(tmp/sum)>1.0e-18);

	return sum;
*/
	double y = x-1.0;
	double tmp = 1.0;
	tmp += y*(k+1.0)/2.0;
	tmp += y*y*(k+1.0)*(k-1.0)/6.0;
	tmp += y*y*y*(k+1.0)*(k-1.0)*(k-2.0)/24.0;
	tmp += y*y*y*y*(k+1.0)*(k-1.0)*(k-2.0)*(k-3.0)/120.0;
	tmp += y*y*y*y*y*(k+1.0)*(k-1.0)*(k-2.0)*(k-3.0)*(k-4.0)/720.0;

	return tmp;
}

//=======================================================================================

__device__ void get_PEM_parameters(int i, int geom, double* x, double* dx, double* dv, double* a, double* par)
{
	double a1, a2, a3, x1, x2, x3, sL, sR, aL, aR;

	x1 = dx[i-1];
	x2 = dx[i];
	x3 = dx[i+1];

	a1 = a[i-1]*dv[i-1]/x1;
	a2 = a[i]*dv[i]/x2;
	a3 = a[i+1]*dv[i+1]/x3;

	//===============================================================================

	sL = (a2-a1)/(x2+x1);
	sR = (a3-a2)/(x3+x2);
    
	aL = x2* ((x3+x2)*sL + x1*sR) / (x1+x2+x3);
	aR = x2* ((x1+x2)*sR + x3*sL) / (x1+x2+x3);

	sL = copysign(fmin(fabs(aL),fabs(a2-a1)),a2-a1);
	sR = copysign(fmin(fabs(aR),fabs(a3-a2)),a3-a2);

	aL = a2 - sL;
	aR = a2 + sR;

	//===============================================================================

	if (sR*sL<=0.0)
	{
		par[0] = 0.0;
		par[1] = a2;
		par[2] = 0.0;
		par[3] = 1.0;
	}
	else
	{
		//if (aR/aL>2.0) aR = 2.0*aL;
		//if (aL/aR>2.0) aL = 2.0*aR;
		par[0] = sL;
		par[1] = a2;
		par[2] = sR;
		par[3] = sR/sL;
    	}
	return;
}

//=======================================================================================

__device__ double lim01(double a)
{
	return fmin(fmax(a,0.0),1.0);
}

__device__ double get_PEM_0(double r0, double rx, double r1, double s0, double aM, double eta)
{
	double dr = r1-r0;
	double x = (rx-r0)/dr;

	double val;
	val  = aM + s0*(1.0-exp(eta*log(x)));

	return val;
}

__device__ double get_PEM_1(double r0, double rx, double r1, double aM, double s1, double eta)
{
	double dr = r1-r0;
	double x = (rx-r0)/dr;
	double y = (r1-rx)/dr;

	double val;
	if (eta*y>1e-3) val = aM + s1*lim01(x*(1.0-exp(eta*log(x)))/(y*eta));
	else            val = aM + s1*get_g_apprx(x, eta);

	return val;
}

//=======================================================================================

__device__ double get_PEM_aveR(int geom, double rL, double r0, double rR, double* par)
{
	double aM = par[1];
	double sR = par[2];
	double eta = par[3];
	double val;

	if (eta>=1.0) val = get_PEM_1(rL, r0, rR, aM, sR, eta);
	else          val = get_PEM_0(rR, r0, rL, sR, aM, 1.0/eta);

	return val/get_dv_dr(geom, r0, rR-r0);
}

__device__ double get_PEM_aveL(int geom, double rL, double r0, double rR, double* par)
{
	double aM = par[1];
	double sL = par[0];
	double eta = par[3];
	double val;

	if (eta>=1.0) val = get_PEM_0(rL, r0, rR, -sL, aM, eta);
	else          val = get_PEM_1(rR, r0, rL, aM, -sL, 1.0/eta);

	return val/get_dv_dr(geom, rL, r0-rL);
}

__device__ double get_PEM_ave(int i, int geom, double* r, double* dr, double* dv, double* a, double c, double dt)
{
	double par[4];
	double rL, rR;

	get_PEM_parameters(i,geom,r,dr,dv,a,par);
	rL = r[i];
	rR = r[i+1];

	if (c*dt>0.0) return get_PEM_aveR(geom, rL, rR-c*dt, rR, par);
	else          return get_PEM_aveL(geom, rL, rL-c*dt, rR, par);
}

